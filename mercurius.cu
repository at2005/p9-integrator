#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#define NUM_BODIES 2
#define MAX_ITERATIONS_ROOT_FINDING 20
#define CUTOFF 1e-13
#define NUM_TIMESTEPS 200
#define G 1
#define TWOPI 6.283185307179586476925286766559005768394338798750211641949

// solves kepler's equation for the eccentric anomaly E
__device__
double danby_burkardt(double mean_anomaly, double eccentricity) {
    // init eccentric anomaly to mean anomaly
    double E = mean_anomaly;
    for(int i = 0; i < MAX_ITERATIONS_ROOT_FINDING; i++) {
        double e_sin = eccentricity * sin(E);
        double f = E - e_sin - mean_anomaly;
        if(fabs(f) < CUTOFF) break;
        double e_cos = eccentricity * cos(E);
        double f_prime = 1 - e_cos; 
        double dE =  - f / f_prime;
        dE = - f / (f_prime + dE*e_sin / 2.00);
        // quartic convergence
        dE = - f / ((f_prime + dE*e_sin / 2.00) + (dE*dE*e_cos / 6.00));
        // quintic convergence
        dE = - f / ((f_prime + dE*e_sin / 2.00) + (dE*dE*e_cos / 6.00) - (dE*dE*dE*e_sin / 24.00));
        E += dE;    
    }

    return E;
}

__device__
double changeover(double r_ij) {
    // double r_crit = 0.001;
    // double y = (r_ij - 0.1*r_crit) / (0.9*r_crit);
    // double K = y*y / (2*y*y - 2*y + 1);
    // return K;
    return 1.0;
}

__device__
void cartesian_from_elements(
    double* vec_inclination, 
    double* vec_longitude_of_ascending_node, 
    double* vec_argument_of_perihelion, 
    double* vec_mean_anomaly,
    double* vec_eccentricity,
    double* vec_semi_major_axis,
    double3* current_positions,
    double3* current_velocities
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    double inclination = vec_inclination[idx];
    double longitude_of_ascending_node = vec_longitude_of_ascending_node[idx];
    double argument_of_perihelion = vec_argument_of_perihelion[idx];
    double mean_anomaly = vec_mean_anomaly[idx];
    double eccentricity = vec_eccentricity[idx];
    double semi_major_axis = vec_semi_major_axis[idx];

    double cos_i = cos(inclination);
    double sin_i = sin(inclination);
    double cos_o = cos(longitude_of_ascending_node);
    double sin_o = sin(longitude_of_ascending_node);
    double cos_a = cos(argument_of_perihelion);
    double sin_a = sin(argument_of_perihelion);
    
    double z1 = cos_a * cos_o;
    double z2 = cos_a * sin_o;
    double z3 = sin_a * cos_o;
    double z4 = sin_a * sin_o;
    double d11 =  z1 - z4*cos_i;
    double d12 =  z2 + z3*cos_i;
    double d13 = sin_a * sin_i;
    double d21 = -z3 - z2*cos_i;
    double d22 = -z4 + z1*cos_i;
    double d23 = cos_a * sin_i;
   
    double romes = sqrt(1 - eccentricity*eccentricity);
    double eccentric_anomaly = danby_burkardt(mean_anomaly, eccentricity);
    double sin_e = sin(eccentric_anomaly);
    double cos_e = cos(eccentric_anomaly);
    z1 = semi_major_axis * (cos_e - eccentricity);
    z2 = semi_major_axis * romes * sin_e;
    eccentric_anomaly = sqrt(G/semi_major_axis) / (1.0 - eccentricity*cos_e);
    z3 = -sin_e * eccentric_anomaly;
    z4 = romes * cos_e * eccentric_anomaly;
    
    current_positions[idx] = make_double3(d11 * z1 + d21 * z2, d12 * z1 + d22 * z2, d13 * z1 + d23 * z2);
    current_velocities[idx] = make_double3(d11 * z3 + d21 * z4, d12 * z3 + d22 * z4, d13 * z3 + d23 * z4);
}


__device__ double3 cross(const double3& a, const double3& b) {
    return make_double3(
        a.y * b.z - a.z * b.y,
        a.z * b.x - a.x * b.z,
        a.x * b.y - a.y * b.x
    );
}

__device__ double magnitude(const double3& a) {
    return sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
}

__device__ double3 magnitude_squared(const double3& a) {
    return make_double3(a.x * a.x, a.y * a.y, a.z * a.z);
}

__device__
void elements_from_cartesian(
    double3* current_positions,
    double3* current_velocities,
    double* vec_inclination, 
    double* vec_longitude_of_ascending_node, 
    double* vec_argument_of_perihelion, 
    double* vec_mean_anomaly,
    double* vec_eccentricity,
    double* vec_semi_major_axis
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    double3 current_p = current_positions[idx];
    double3 current_v = current_velocities[idx];
    double3 angular_momentum = cross(current_p, current_v);
    double epsilon = 1e-8;
    double h_sq = magnitude_squared(angular_momentum).x + magnitude_squared(angular_momentum).y + magnitude_squared(angular_momentum).z + epsilon;
    double inclination = acos(angular_momentum.z / sqrt(h_sq));
    double longitude_of_ascending_node = atan2(angular_momentum.x, -angular_momentum.y == 0.0 ? 0.0 : -angular_momentum.y);
    double v_sq = magnitude_squared(current_v).x + magnitude_squared(current_v).y + magnitude_squared(current_v).z;
    double r = magnitude(current_p);
    double s = h_sq / G;
    double eccentricity = sqrt(1 + s * ((v_sq / G) - (2.00 / r)));
    double perihelion_distance = s / (1.00 + eccentricity);
    double cos_e = (v_sq*r - G) / (eccentricity*G);
    double E_anomaly = acos(cos_e);
    double M_anomaly = E_anomaly - eccentricity * sin(E_anomaly); 
    double cos_f = (s - r ) / (eccentricity * r);
    double f = acos(cos_f);

    double to = -angular_momentum.x / angular_momentum.y;
    double temp = (1.00 - cos(inclination)) * to;
    double temp2 = to * to;
    double true_longitude = atan2((current_p.y * (1.00 + temp2 * cos(inclination)) - current_p.x * temp), (current_p.x * (temp2 + cos(inclination)) - current_p.y * temp));

    double p = true_longitude - f;
    p = fmod(p + TWOPI + TWOPI, TWOPI);
    double argument_of_perihelion = p - longitude_of_ascending_node;
    double semi_major_axis = perihelion_distance / (1.00 - eccentricity);

    vec_inclination[idx] = inclination;
    vec_longitude_of_ascending_node[idx] = longitude_of_ascending_node;
    vec_argument_of_perihelion[idx] = argument_of_perihelion;
    vec_mean_anomaly[idx] = M_anomaly;
    vec_eccentricity[idx] = eccentricity;
    vec_semi_major_axis[idx] = semi_major_axis;
}

__device__
void body_interaction_kick(double3* positions, double3* velocities, double* masses, double dt) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    double3 acc = make_double3(0.0, 0.0, 0.0);
    double dist_x, dist_y, dist_z = 0.0;
    for(int i = 0; i < NUM_BODIES; i++) {
        if(i == idx) continue;
       // 3-vec displacement, let r = x, y, z, this is the direction of the acceleration
        dist_x = positions[i].x - positions[idx].x;
        dist_y = positions[i].y - positions[idx].y;
        dist_z = positions[i].z - positions[idx].z;
        double epsilon = 1e-8;
        double r = sqrt(dist_x * dist_x + dist_y * dist_y + dist_z * dist_z);
        // // magnitude of acceleration = mass_of_other_body * G / |r|^3
        double weighted_acceleration = changeover(r) * masses[i] * G / pow(r + epsilon, 3);
        // // accumulate total acceleration due to all bodies, except self
        acc.x += weighted_acceleration * dist_x;
        acc.y += weighted_acceleration * dist_y;
        acc.z += weighted_acceleration * dist_z;
    }

    // update momenta (velocity here) with total acceleration
    velocities[idx].x += acc.x * dt;
    velocities[idx].y += acc.y * dt;
    velocities[idx].z += acc.z * dt;     
}

__device__
void main_body_kinetic(double3* positions, double3* velocities, double* masses, double dt) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    double3 p = make_double3(0.0, 0.0, 0.0);
    // calculate total momentum of all bodies
    for(int i = 1; i < blockDim.x + 1; i++) {
        p.x += velocities[i-1].x * masses[i];
        p.y += velocities[i-1].y * masses[i];
        p.z += velocities[i-1].z * masses[i];
    }

    double scaling_factor = dt/(masses[0]);
    positions[idx].x += p.x * scaling_factor;
    positions[idx].y += p.y * scaling_factor;
    positions[idx].z += p.z * scaling_factor;
}


__device__
void convert_to_democratic_heliocentric_coordinates(double3* positions, double3* velocities, double* masses) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    double total_mass = 0.0;
    double3 mass_weighted_v = make_double3(0.0, 0.0, 0.0);
    for(int i = 0; i < blockDim.x; i++) {
        total_mass += masses[i + 1];
        mass_weighted_v.x += masses[i + 1] * velocities[i].x;
        mass_weighted_v.y += masses[i + 1] * velocities[i].y;
        mass_weighted_v.z += masses[i + 1] * velocities[i].z;
    }

    double scaling_factor = 1.00 / (total_mass + masses[0]);
    mass_weighted_v.x *= scaling_factor;
    mass_weighted_v.y *= scaling_factor;
    mass_weighted_v.z *= scaling_factor;

    velocities[idx].x -= mass_weighted_v.x;
    velocities[idx].y -= mass_weighted_v.y;
    velocities[idx].z -= mass_weighted_v.z;
    
}

__global__ 
void mercurius_keplerian_solver(
    double* vec_argument_of_perihelion_hbm,
    double* vec_mean_anomaly_hbm,
    double* vec_eccentricity_hbm,
    double* vec_semi_major_axis_hbm,
    double* vec_inclination_hbm,
    double* vec_longitude_of_ascending_node_hbm,
    double* vec_masses_hbm,
    double dt,
    double3* output_positions
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // declare buffers for positions in SRAM
    __shared__ double3 positions[NUM_BODIES];  
    __shared__ double3 velocities[NUM_BODIES];
    __shared__ double masses[NUM_BODIES + 1];
    __shared__ double vec_inclination[NUM_BODIES];
    __shared__ double vec_longitude_of_ascending_node[NUM_BODIES];
    __shared__ double vec_argument_of_perihelion[NUM_BODIES];
    __shared__ double vec_mean_anomaly[NUM_BODIES];
    __shared__ double vec_eccentricity[NUM_BODIES];
    __shared__ double vec_semi_major_axis[NUM_BODIES];

    // copy data to shared memory
    // special case to avoid race condition
    if (idx == 0) masses[0] = vec_masses_hbm[0];

    masses[idx+1] = vec_masses_hbm[idx+1]; 
    vec_argument_of_perihelion[idx] = vec_argument_of_perihelion_hbm[idx];
    vec_mean_anomaly[idx] = vec_mean_anomaly_hbm[idx];
    vec_eccentricity[idx] = vec_eccentricity_hbm[idx];
    vec_semi_major_axis[idx] = vec_semi_major_axis_hbm[idx];
    vec_inclination[idx] = vec_inclination_hbm[idx];
    vec_longitude_of_ascending_node[idx] = vec_longitude_of_ascending_node_hbm[idx];
    __syncthreads(); 
    // initially populate positions and velocities
    cartesian_from_elements(
        vec_inclination,
        vec_longitude_of_ascending_node,
        vec_argument_of_perihelion,
        vec_mean_anomaly,
        vec_eccentricity,
        vec_semi_major_axis,
        positions,
        velocities
    );

    __syncthreads();
    // convert to democratic heliocentric coordinates
    convert_to_democratic_heliocentric_coordinates(positions, velocities, masses);

    for(int i = 0; i < NUM_TIMESTEPS; i++) {
        __syncthreads();
        body_interaction_kick(positions, velocities, masses, dt/2.00);
        __syncthreads();
        main_body_kinetic(positions, velocities, masses, dt/2.00);
        double semi_major_axis = vec_semi_major_axis[idx];
        double n = 1.00 / (semi_major_axis * semi_major_axis * semi_major_axis); 
        __syncthreads(); 
        elements_from_cartesian(
            positions,
            velocities,
            vec_inclination,
            vec_longitude_of_ascending_node,
            vec_argument_of_perihelion,
            vec_mean_anomaly,
            vec_eccentricity,
            vec_semi_major_axis
        );
        __syncthreads();
        // // advance mean anomaly, this is essentially advancing to the next timestep
        vec_mean_anomaly[idx] = fmod(n * dt + vec_mean_anomaly[idx], TWOPI);
        __syncthreads();
        cartesian_from_elements(
            vec_inclination,
            vec_longitude_of_ascending_node,
            vec_argument_of_perihelion,
            vec_mean_anomaly,
            vec_eccentricity,
            vec_semi_major_axis,
            positions,
            velocities
        );

        __syncthreads(); 
        main_body_kinetic(positions, velocities, masses, dt/2.00);
        __syncthreads();
        body_interaction_kick(positions, velocities, masses, dt/2.00);

        // basically the layout here is:
        // [[body0, body1, body2, ...], [body0, body1, body2, ...], ...]
        // where each subarray is a timestep
        // so we need to index into the timestep and then add idx to index a particular body
        output_positions[i* blockDim.x + idx] = positions[idx];
        __syncthreads();
    }
    
}


struct Body {
    double inclination;
    double longitude_of_ascending_node;
    double argument_of_perihelion;
    double mean_anomaly;
    double eccentricity;
    double semi_major_axis;
    double mass;
};

struct Sim {
    int num_bodies;
    double3* positions;
    double3* velocities;
    double* masses;
    double* vec_inclination;
    double* vec_longitude_of_ascending_node;
    double* vec_argument_of_perihelion;
    double* vec_mean_anomaly;
    double* vec_eccentricity;
    double* vec_semi_major_axis;
};

__host__
void initialize_std_sim(Sim* sim, int num_bodies) {
    sim->vec_inclination = (double*)malloc(num_bodies * sizeof(double));
    sim->vec_longitude_of_ascending_node = (double*)malloc(num_bodies * sizeof(double));
    sim->vec_argument_of_perihelion = (double*)malloc(num_bodies * sizeof(double));
    sim->vec_mean_anomaly = (double*)malloc(num_bodies * sizeof(double));
    sim->vec_eccentricity = (double*)malloc(num_bodies * sizeof(double));
    sim->vec_semi_major_axis = (double*)malloc(num_bodies * sizeof(double));
    sim->masses = (double*)malloc((num_bodies+1) * sizeof(double));

    // assume convention that main body mass is 1
    sim->masses[0] = 1.0;
    sim->num_bodies = num_bodies;
}

__host__
void add_body_to_sim(Sim* sim, Body body, int idx) {
    sim->vec_inclination[idx] = body.inclination;
    sim->vec_longitude_of_ascending_node[idx] = body.longitude_of_ascending_node;
    sim->vec_argument_of_perihelion[idx] = body.argument_of_perihelion;
    sim->vec_mean_anomaly[idx] = body.mean_anomaly;
    sim->vec_eccentricity[idx] = body.eccentricity;
    sim->vec_semi_major_axis[idx] = body.semi_major_axis;
    sim->masses[idx+1] = body.mass;
}


void dump_sim(Sim* sim) {
    std::cout << "Simulation with " << sim->num_bodies << " bodies" << std::endl;
    std::cout << "Main body mass: " << sim->masses[0] << std::endl;
    for(int i = 0; i < sim->num_bodies; i++) {
        std::cout << "Body: " << i << std::endl;
        std::cout << "inclination: " << sim->vec_inclination[i] << std::endl;
        std::cout << "longitude of ascending node: " << sim->vec_longitude_of_ascending_node[i] << std::endl;
        std::cout << "argument of perihelion: " << sim->vec_argument_of_perihelion[i] << std::endl;
        std::cout << "mean anomaly: " << sim->vec_mean_anomaly[i] << std::endl;
        std::cout << "eccentricity: " << sim->vec_eccentricity[i] << std::endl;
        std::cout << "semi major axis: " << sim->vec_semi_major_axis[i] << std::endl;
        std::cout << "mass: " << sim->masses[i+1] << std::endl << std::endl;
    }
}

__host__
int main() {
    Sim sim;
    initialize_std_sim(&sim, NUM_BODIES);
    double dt = 0.5;

    // testing 3-body system
    Body Earth;
    Earth.inclination = 0.00005 * M_PI / 180.0;
    Earth.longitude_of_ascending_node = -11.26064 * M_PI / 180.0;
    Earth.argument_of_perihelion = 102.94719 * M_PI / 180.0;
    Earth.mean_anomaly = 100.46435 * M_PI / 180.0;
    Earth.eccentricity = 0.01671022;
    Earth.semi_major_axis = 1.00000011;
    Earth.mass = 5.97237e24 / 1.98855e30;

    // add earth to simulation
    add_body_to_sim(&sim, Earth, 0);

    Body Mars;
    Mars.inclination = 1.848 * M_PI / 180.0;
    Mars.longitude_of_ascending_node = 49.57854 * M_PI / 180.0;
    Mars.argument_of_perihelion = 336.04084 * M_PI / 180.0;
    Mars.mean_anomaly = 0;
    Mars.eccentricity = 0.0934;
    Mars.semi_major_axis = 1.5;
    Mars.mass = 0.000954588;

    // yay now we add mars to the simulation
    add_body_to_sim(&sim, Mars, 1);

    // print sim information 
    dump_sim(&sim);

    // this is bc we need to allocate memory on the device
    double *vec_longitude_of_ascending_node_device, *vec_inclination_device, *vec_argument_of_perihelion_device, 
        *vec_mean_anomaly_device, *vec_eccentricity_device, *vec_semi_major_axis_device, *masses_device;
    double3 *output_positions_device;
    double3* output_positions = (double3*)malloc(sim.num_bodies * sizeof(double3) * NUM_TIMESTEPS);

    hipMalloc((void**)&vec_longitude_of_ascending_node_device, sim.num_bodies * sizeof(double));
    hipMalloc((void**)&vec_inclination_device, sim.num_bodies * sizeof(double));
    hipMalloc((void**)&vec_argument_of_perihelion_device, sim.num_bodies * sizeof(double));
    hipMalloc((void**)&vec_mean_anomaly_device, sim.num_bodies * sizeof(double));
    hipMalloc((void**)&vec_eccentricity_device, sim.num_bodies * sizeof(double));
    hipMalloc((void**)&vec_semi_major_axis_device, sim.num_bodies * sizeof(double));
    hipMalloc((void**)&masses_device, (sim.num_bodies + 1) * sizeof(double));
    hipMalloc((void**)&output_positions_device, sim.num_bodies * sizeof(double3) * NUM_TIMESTEPS);

    hipMemcpy(vec_longitude_of_ascending_node_device, sim.vec_longitude_of_ascending_node, sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vec_inclination_device, sim.vec_inclination, sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vec_argument_of_perihelion_device, sim.vec_argument_of_perihelion, sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vec_mean_anomaly_device, sim.vec_mean_anomaly, sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vec_eccentricity_device, sim.vec_eccentricity, sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vec_semi_major_axis_device, sim.vec_semi_major_axis, sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(masses_device, sim.masses, (sim.num_bodies+1) * sizeof(double), hipMemcpyHostToDevice);

    std::cout << "Launching kernel on " << sim.num_bodies << " threads" << std::endl;
    mercurius_keplerian_solver<<<1, sim.num_bodies>>>(
        vec_argument_of_perihelion_device,
        vec_mean_anomaly_device,
        vec_eccentricity_device,
        vec_semi_major_axis_device,
        vec_inclination_device,
        vec_longitude_of_ascending_node_device,
        masses_device,
        dt,
        output_positions_device
    );

    std::cout << "Synchronizing...\n";
    hipDeviceSynchronize();
    hipMemcpy(output_positions, output_positions_device, sim.num_bodies * sizeof(double3) * NUM_TIMESTEPS, hipMemcpyDeviceToHost);

    // print output positions
    std::cout << "Output positions:" << std::endl;
    for(int i = 0; i < NUM_TIMESTEPS; i++) {
        std::cout << "Timestep " << i << std::endl;
        for(int j = 0; j < sim.num_bodies; j++) {
            std::cout << output_positions[i*sim.num_bodies + j].x << " " << output_positions[i*sim.num_bodies + j].y << " " << output_positions[i*sim.num_bodies + j].z << std::endl;
        }
    }

    hipFree(vec_longitude_of_ascending_node_device);
    hipFree(vec_inclination_device);
    hipFree(vec_argument_of_perihelion_device);
    hipFree(vec_mean_anomaly_device);
    hipFree(vec_eccentricity_device);
    hipFree(vec_semi_major_axis_device);
    hipFree(masses_device);
    hipFree(output_positions_device);
    hipDeviceReset();    
}
 