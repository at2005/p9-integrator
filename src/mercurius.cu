#include "hip/hip_runtime.h"
#include "sim.cuh"
#include "simutils.cuh"

__host__ int main(int argc, char **argv)
{
  // cli args
  bool print_sim_info = false;
  bool print_positions = false;
  std::string output_file;
  std::string config_file;
  // default is one orbital period (of Earth)
  int NUM_TIMESTEPS = 1;
  args_parse(argc,
             argv,
             &print_sim_info,
             &print_positions,
             &NUM_TIMESTEPS,
             &config_file,
             &output_file);

  Sim sim;
  sim_from_config_file(&sim, config_file, NUM_TIMESTEPS);

  // set integration timestep to the one BB21 use
  double dt = 0.8219;
  // double dt = 0.1;

  // this is bc we need to allocate memory on the device (on HBM – global
  // memory, copy to SRAM later)
  double *vec_longitude_of_ascending_node_device, *vec_inclination_device,
      *vec_argument_of_perihelion_device, *vec_mean_anomaly_device,
      *vec_eccentricity_device, *vec_semi_major_axis_device, *masses_device;
  double3 *output_positions_device;
  double3 *output_positions =
      (double3 *)malloc(sim.num_bodies * sizeof(double3) * BATCH_SIZE);

  hipMalloc((void **)&vec_longitude_of_ascending_node_device,
             sim.num_bodies * sizeof(double));
  hipMalloc((void **)&vec_inclination_device, sim.num_bodies * sizeof(double));
  hipMalloc((void **)&vec_argument_of_perihelion_device,
             sim.num_bodies * sizeof(double));
  hipMalloc((void **)&vec_mean_anomaly_device,
             sim.num_bodies * sizeof(double));
  hipMalloc((void **)&vec_eccentricity_device,
             sim.num_bodies * sizeof(double));
  hipMalloc((void **)&vec_semi_major_axis_device,
             sim.num_bodies * sizeof(double));
  hipMalloc((void **)&masses_device, sim.num_bodies * sizeof(double));
  hipMalloc((void **)&output_positions_device,
             sim.num_bodies * sizeof(double3) * BATCH_SIZE);

  hipMemcpy(vec_longitude_of_ascending_node_device,
             sim.vec_longitude_of_ascending_node,
             sim.num_bodies * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(vec_inclination_device,
             sim.vec_inclination,
             sim.num_bodies * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(vec_argument_of_perihelion_device,
             sim.vec_argument_of_perihelion,
             sim.num_bodies * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(vec_mean_anomaly_device,
             sim.vec_mean_anomaly,
             sim.num_bodies * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(vec_eccentricity_device,
             sim.vec_eccentricity,
             sim.num_bodies * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(vec_semi_major_axis_device,
             sim.vec_semi_major_axis,
             sim.num_bodies * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(masses_device,
             sim.masses,
             sim.num_bodies * sizeof(double),
             hipMemcpyHostToDevice);

  // print sim information
  if (print_sim_info)
  {
    dump_sim(&sim);
    std::cout << "Launching kernel on " << sim.num_bodies << " threads"
              << std::endl;
  }

  // positions and velocity 3-vectors, 6 orbital elements for each body, mass
  // for each body (so 7 doubles)
  size_t sram_size = sim.num_bodies * sizeof(double3) * 2 +
                     sim.num_bodies * sizeof(double) * 7;

  if (print_sim_info) std::cout << "Allocating " << sram_size << " bytes of SRAM" << std::endl;

  // ie after BATCH_SIZE timesteps, we want to print the output
  // and run kernel with updated orbital elements this is to save memory
  int NUM_ITERS = NUM_TIMESTEPS > BATCH_SIZE ? NUM_TIMESTEPS / BATCH_SIZE : NUM_TIMESTEPS;
  if (NUM_TIMESTEPS > BATCH_SIZE) assert(NUM_TIMESTEPS % BATCH_SIZE == 0);

  size_t max_sram = 227 * 1024;
  hipFuncSetAttribute(reinterpret_cast<const void*>(mercurius_solver), hipFuncAttributeMaxDynamicSharedMemorySize, max_sram);

  for (int batch = 0; batch < NUM_ITERS; batch++)
  {
    mercurius_solver<<<1, sim.num_bodies, sram_size>>>(
        vec_argument_of_perihelion_device,
        vec_mean_anomaly_device,
        vec_eccentricity_device,
        vec_semi_major_axis_device,
        vec_inclination_device,
        vec_longitude_of_ascending_node_device,
        masses_device,
        output_positions_device,
        dt);

    if (print_sim_info) std::cout << "Batch " << (batch + 1) << " Simulation Complete. Synchronizing...\n";
    hipDeviceSynchronize();
    hipMemcpy(output_positions,
               output_positions_device,
               sim.num_bodies * sizeof(double3) * BATCH_SIZE,
               hipMemcpyDeviceToHost);

    if (print_positions) pretty_print_positions(&sim, output_positions, batch);
    // if(output_file != "") write_output(output_positions, batch, output_file);
  }

  hipFree(vec_longitude_of_ascending_node_device);
  hipFree(vec_inclination_device);
  hipFree(vec_argument_of_perihelion_device);
  hipFree(vec_mean_anomaly_device);
  hipFree(vec_eccentricity_device);
  hipFree(vec_semi_major_axis_device);
  hipFree(masses_device);
  hipFree(output_positions_device);
  hipDeviceReset();
}
