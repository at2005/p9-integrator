#include "hip/hip_runtime.h"
#include "simutils.cuh"
#include "sim.cuh"

__host__
int main(int argc, char** argv) {
        // cli args
    bool print_sim_info = false;
    bool print_positions = false;
    // default is one orbital period (of Earth)
    int NUM_TIMESTEPS = 1;
    args_parse(argc, argv, &print_sim_info, &print_positions, &NUM_TIMESTEPS);

    Sim sim;
    initialize_std_sim(&sim, NUM_BODIES, NUM_TIMESTEPS);
    double dt = 0.08;

    // testing 3-body system
    Body Earth;
    Earth.inclination = 0.00005 * M_PI / 180.0;
    Earth.longitude_of_ascending_node = -11.26064 * M_PI / 180.0;
    Earth.argument_of_perihelion = 102.94719 * M_PI / 180.0;
    Earth.mean_anomaly = 100.46435 * M_PI / 180.0;
    Earth.eccentricity = 0.01671022;
    Earth.semi_major_axis = 1.00000011;
    Earth.mass = 5.97237e24 / 1.98855e30;
    Earth.name = "Earth";

    // add earth to simulation
    add_body_to_sim(&sim, Earth, 0);

    Body Mars;
    Mars.inclination = 1.848 * M_PI / 180.0;
    Mars.longitude_of_ascending_node = 49.57854 * M_PI / 180.0;
    Mars.argument_of_perihelion = 336.04084 * M_PI / 180.0;
    Mars.mean_anomaly = 0;
    Mars.eccentricity = 0.0934;
    Mars.semi_major_axis = 1.5;
    Mars.mass = 0.000954588;
    Mars.name = "Mars";

    // yay now we add mars to the simulation
    add_body_to_sim(&sim, Mars, 1);

    // this is bc we need to allocate memory on the device (on HBM – global memory, copy to SRAM later)
    double *vec_longitude_of_ascending_node_device, *vec_inclination_device, *vec_argument_of_perihelion_device, 
        *vec_mean_anomaly_device, *vec_eccentricity_device, *vec_semi_major_axis_device, *masses_device;
    double3 *output_positions_device;
    double3* output_positions = (double3*)malloc(sim.num_bodies * sizeof(double3) * NUM_TIMESTEPS);

    hipMalloc((void**)&vec_longitude_of_ascending_node_device, sim.num_bodies * sizeof(double));
    hipMalloc((void**)&vec_inclination_device, sim.num_bodies * sizeof(double));
    hipMalloc((void**)&vec_argument_of_perihelion_device, sim.num_bodies * sizeof(double));
    hipMalloc((void**)&vec_mean_anomaly_device, sim.num_bodies * sizeof(double));
    hipMalloc((void**)&vec_eccentricity_device, sim.num_bodies * sizeof(double));
    hipMalloc((void**)&vec_semi_major_axis_device, sim.num_bodies * sizeof(double));
    hipMalloc((void**)&masses_device, (sim.num_bodies + 1) * sizeof(double));
    hipMalloc((void**)&output_positions_device, sim.num_bodies * sizeof(double3) * NUM_TIMESTEPS);

    hipMemcpy(vec_longitude_of_ascending_node_device, sim.vec_longitude_of_ascending_node, sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vec_inclination_device, sim.vec_inclination, sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vec_argument_of_perihelion_device, sim.vec_argument_of_perihelion, sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vec_mean_anomaly_device, sim.vec_mean_anomaly, sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vec_eccentricity_device, sim.vec_eccentricity, sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vec_semi_major_axis_device, sim.vec_semi_major_axis, sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(masses_device, sim.masses, (sim.num_bodies+1) * sizeof(double), hipMemcpyHostToDevice);


    // print sim information 
    if(print_sim_info) {
        dump_sim(&sim);
        std::cout << "Launching kernel on " << sim.num_bodies << " threads" << std::endl;
    }

    mercurius_keplerian_solver<<<1, sim.num_bodies>>>(
        vec_argument_of_perihelion_device,
        vec_mean_anomaly_device,
        vec_eccentricity_device,
        vec_semi_major_axis_device,
        vec_inclination_device,
        vec_longitude_of_ascending_node_device,
        masses_device,
        output_positions_device,
        dt,
       NUM_TIMESTEPS
    );

    if(print_sim_info) std::cout << "Simulation Finished. Synchronizing...\n";
    hipDeviceSynchronize();
    hipMemcpy(output_positions, output_positions_device, sim.num_bodies * sizeof(double3) * NUM_TIMESTEPS, hipMemcpyDeviceToHost);
    
   if(print_positions) pretty_print_positions(&sim, output_positions);

    hipFree(vec_longitude_of_ascending_node_device);
    hipFree(vec_inclination_device);
    hipFree(vec_argument_of_perihelion_device);
    hipFree(vec_mean_anomaly_device);
    hipFree(vec_eccentricity_device);
    hipFree(vec_semi_major_axis_device);
    hipFree(masses_device);
    hipFree(output_positions_device);
    hipDeviceReset();    
}
 