#include "hip/hip_runtime.h"
#include "sim.cuh"
#include "simutils.cuh"

__host__ int main(int argc, char **argv) {
  // cli args
  bool print_sim_info = false;
  bool print_positions = false;
  std::string config_file;
  // default is one orbital period (of Earth)
  int NUM_TIMESTEPS = 1;
  args_parse(argc, argv, &print_sim_info, &print_positions, &NUM_TIMESTEPS,
             &config_file);

  Sim sim;
  sim_from_config_file(&sim, config_file, NUM_TIMESTEPS);

  // set integration timestep to the one BB21 use
  double dt = 0.8219;

  // this is bc we need to allocate memory on the device (on HBM – global
  // memory, copy to SRAM later)
  double *vec_longitude_of_ascending_node_device, *vec_inclination_device,
      *vec_argument_of_perihelion_device, *vec_mean_anomaly_device,
      *vec_eccentricity_device, *vec_semi_major_axis_device, *masses_device;
  double3 *output_positions_device;
  double3 *output_positions =
      (double3 *)malloc(sim.num_bodies * sizeof(double3) * NUM_TIMESTEPS);

  hipMalloc((void **)&vec_longitude_of_ascending_node_device,
             sim.num_bodies * sizeof(double));
  hipMalloc((void **)&vec_inclination_device, sim.num_bodies * sizeof(double));
  hipMalloc((void **)&vec_argument_of_perihelion_device,
             sim.num_bodies * sizeof(double));
  hipMalloc((void **)&vec_mean_anomaly_device,
             sim.num_bodies * sizeof(double));
  hipMalloc((void **)&vec_eccentricity_device,
             sim.num_bodies * sizeof(double));
  hipMalloc((void **)&vec_semi_major_axis_device,
             sim.num_bodies * sizeof(double));
  hipMalloc((void **)&masses_device, (sim.num_bodies + 1) * sizeof(double));
  hipMalloc((void **)&output_positions_device,
             sim.num_bodies * sizeof(double3) * NUM_TIMESTEPS);

  hipMemcpy(vec_longitude_of_ascending_node_device,
             sim.vec_longitude_of_ascending_node,
             sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(vec_inclination_device, sim.vec_inclination,
             sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(vec_argument_of_perihelion_device, sim.vec_argument_of_perihelion,
             sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(vec_mean_anomaly_device, sim.vec_mean_anomaly,
             sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(vec_eccentricity_device, sim.vec_eccentricity,
             sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(vec_semi_major_axis_device, sim.vec_semi_major_axis,
             sim.num_bodies * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(masses_device, sim.masses, (sim.num_bodies + 1) * sizeof(double),
             hipMemcpyHostToDevice);

  // print sim information
  if (print_sim_info) {
    dump_sim(&sim);
    std::cout << "Launching kernel on " << sim.num_bodies << " threads"
              << std::endl;
  }

  mercurius_solver<<<1, sim.num_bodies>>>(
      vec_argument_of_perihelion_device, vec_mean_anomaly_device,
      vec_eccentricity_device, vec_semi_major_axis_device,
      vec_inclination_device, vec_longitude_of_ascending_node_device,
      masses_device, output_positions_device, dt, NUM_TIMESTEPS);

  if (print_sim_info)
    std::cout << "Simulation Finished. Synchronizing...\n";
  hipDeviceSynchronize();
  hipMemcpy(output_positions, output_positions_device,
             sim.num_bodies * sizeof(double3) * NUM_TIMESTEPS,
             hipMemcpyDeviceToHost);

  if (print_positions)
    pretty_print_positions(&sim, output_positions);

  hipFree(vec_longitude_of_ascending_node_device);
  hipFree(vec_inclination_device);
  hipFree(vec_argument_of_perihelion_device);
  hipFree(vec_mean_anomaly_device);
  hipFree(vec_eccentricity_device);
  hipFree(vec_semi_major_axis_device);
  hipFree(masses_device);
  hipFree(output_positions_device);
  hipDeviceReset();
}
